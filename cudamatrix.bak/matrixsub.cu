#include "hip/hip_runtime.h"
#include "matrix.h"
#include "externs.h"

// ��������� ���� ������ � �������������� global ������
template<class T> __global__ void __global__matrix_sub__global__(T *a, T *b, T *c, int height, int width, size_t pitch1, size_t pitch2, size_t pitch3) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			ELEMENT(T,c,i,j,pitch3) = ELEMENT(T,a,i,j,pitch1) - ELEMENT(T,b,i,j,pitch2);
		}
	}
}

// ��������� ���� ������ � �������������� constant ������
template<class T> __global__ void __global__matrix_sub__constant__(T *c, int height, int width, size_t pitch) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			ELEMENT(T,c,i,j,pitch) = ELEMENT(T,__c__,i,j,width*sizeof(T)) - ELEMENT(T,__c__,height+i,j,width*sizeof(T));
		}
	}
}

// ��������� ���� ������ � �������������� texture ������
template<class T> __global__ void __global__matrix_sub__texture__(T *c, int height, int width, size_t pitch) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			uint4 buffer_a = tex1Dfetch(tex_a,IDX(i,j,width));
			uint4 buffer_b = tex1Dfetch(tex_b,IDX(i,j,width));
			ELEMENT(T,c,i,j,pitch) = *(T*)&buffer_a - *(T*)&buffer_b;
		}
	}
}

template<class T> __host__ void __host__matrix_sub(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, MATRIX<T> *c, MEMORY src, MEMORY dest, MEMORY cache) {
	T * d_a;
	T * d_b;
	T * d_c;
	size_t pitch1;
	size_t pitch2;
	size_t pitch3;
	hipError_t err;

	int height = a->height;
	int width =  b->width;
	c->height = height;
	c->width = width;

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipHostMalloc((void**)&cpu_b, (size_t) b->width * b->height * sizeof(uint4));
		err = hipMalloc((void**)&gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipMalloc((void**)&gpu_b, (size_t) b->width * b->height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				uint4 buffer;
				*(T*)&buffer = a->values[IDX(i,j,width)];
				cpu_a[IDX(i,j,width)] = buffer;
			}
		}
		for(int i=0; i<b->height; i++) {
			for(int j=0; j<b->width; j++) {
				uint4 buffer;
				*(T*)&buffer = b->values[IDX(i,j,width)];
				cpu_b[IDX(i,j,width)] = buffer;
			}
		}
		//  ��������� ���������� ��������  texture
		tex_a.addressMode[0] = hipAddressModeWrap;  // ����� Wrap
		tex_a.addressMode[1] = hipAddressModeWrap;
		tex_a.filterMode     = hipFilterModePoint;  // ��������� ��������
		tex_a.normalized     = false;                // �� ������������ ��������������� ���������
		tex_b.addressMode[0] = hipAddressModeWrap;  // ����� Wrap
		tex_b.addressMode[1] = hipAddressModeWrap;
		tex_b.filterMode     = hipFilterModePoint;  // ��������� ��������
		tex_b.normalized     = false;                // �� ������������ ��������������� ���������
		err = hipBindTexture(0, tex_a, gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipBindTexture(0, tex_b, gpu_b, (size_t) b->width * b->height * sizeof(uint4));
		err = hipMemcpy((void*)gpu_a, (void*)cpu_a, (size_t) a->width * a->height * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipMemcpy((void*)gpu_b, (void*)cpu_b, (size_t) b->width * b->height * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipHostFree((void*)cpu_a);
		err = hipHostFree((void*)cpu_b);
		break;
	case GLOBAL:
		err = hipMallocPitch((void**)&d_a, &pitch1, (size_t) a->width * sizeof(T), (size_t) a->height);
		err = hipMallocPitch((void**)&d_b, &pitch2, (size_t) b->width * sizeof(T), (size_t) b->height);
		err = hipMemcpy2D((void*)d_a, pitch1, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		err = hipMemcpy2D((void*)d_b, pitch2, (void*)b->values, (size_t) b->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		break;
	case CONSTANT:
		err = hipMemcpyToSymbol(HIP_SYMBOL(__c__), (void*)a->values, (size_t) a->height * a->width * sizeof(T), (size_t) 0, hipMemcpyHostToDevice);
		err = hipMemcpyToSymbol(HIP_SYMBOL(__c__), (void*)b->values, (size_t) b->height * b->width * sizeof(T), (size_t) height*width*sizeof(T), hipMemcpyHostToDevice);
		break;
	default:
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMallocPitch((void**)&d_c, &pitch3, (size_t) c->width * sizeof(T), (size_t) c->height);
		break;
	default:
		break;
	}

	if (src == GLOBAL && dest == GLOBAL) {
		__global__matrix_sub__global__<T><<<blocks,threads>>>(d_a, d_b, d_c, height, width, pitch1, pitch2, pitch3);
	}
	else if (src == CONSTANT && dest == GLOBAL) {
		__global__matrix_sub__constant__<T><<<blocks,threads>>>(d_c, height, width, pitch3);
	}
	else if (src == TEXTURE && dest == GLOBAL) {
		__global__matrix_sub__texture__<T><<<blocks,threads>>>(d_c, height, width, pitch3);
	}

	switch(src) {
	case TEXTURE:
		err = hipUnbindTexture(tex_a);
		err = hipUnbindTexture(tex_b);
		err = hipFree(gpu_a);
		err = hipFree(gpu_b);
		break;
	case CONSTANT:
		break;
	default:
		err = hipFree(d_a);
		err = hipFree(d_b);
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMemcpy2D((void*)c->values, (size_t) c->width * sizeof(T), (void*)d_c, pitch3, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
		err = hipFree(d_c);
		break;
	default:
		break;
	}
	
	err = err;
}
