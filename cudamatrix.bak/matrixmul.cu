#include "hip/hip_runtime.h"
﻿#include "matrix.h"
#include "externs.h"

// Умножение двух матриц с использованием global памяти
template<class T> __global__ void __global__matrix_mul__global__none__(T *a, T *b, T *c, int height, int width_height, int width, size_t pitch1, size_t pitch2, size_t pitch3) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			T sum = 0;
			for (int k = 0; k < width_height ; k++ ) {
				sum += ELEMENT(T,a,i,k,pitch1) * ELEMENT(T,b,k,j,pitch2);
			}
			ELEMENT(T,c,i,j,pitch3) = sum;
		}
	}
}

// Умножение двух матриц с использованием shared памяти
// Using shared memory
// • Similar to local memory:
// – __shared__ float current_row[];
// • Only declare one variable as shared!
// – Multiple declarations of __shared__
// variables will occupy same memory space!
// – __shared__ float a[];
// – __shared__ float b[];
// – b[0] = 0.5f;
// – now a[0] == 0.5f also!
template<class T> __global__ void __global__matrix_mul__global__shared__(T *a, T *b, T *c, int height,int width_height, int width, size_t pitch1, size_t pitch2, size_t pitch3) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	
	__shared__ T __s__[SHARED_SIZE];
	
	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
			__s__[IDX(i,j,width_height)] = ELEMENT(T,a,sharedIndexX+i,j,pitch1);
		}
	}

	for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = ELEMENT(T,b,i,sharedIndexY+j,pitch2);
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
			}
			ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch3) = sum;
		}
	}
}

// Умножение двух матриц с использованием local памяти
// Local memory
// • Also read/write per-thread
// • Can’t read other threads’ local memory
// – Declare a variable in local memory using
// the __local__ keyword
// – __local__ float results[32];
// • Can index (this is where local arrays go)
// • Much slower than register memory!
// – Don’t use local arrays if you don’t have to
template<class T> __global__ void __global__matrix_mul__global__local__(T *a, T *b, T *c, int height,int width_height, int width, size_t pitch1, size_t pitch2, size_t pitch3) {
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x);
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y);
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x)-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/blockDim.x/gridDim.x);
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y)-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/blockDim.y/gridDim.y);
	
	T __l__[LOCAL_SIZE];
	
	for (int i = 0; i < localCountX; i ++) {
		for (int j = 0; j < width_height; j++) {
			__l__[IDX(i,j,width_height)] = ELEMENT(T,a,localIndexX+i,j,pitch1);
		}
	}

	for (int i = 0; i < width_height; i ++) {
		for (int j = 0; j < localCountY; j++) {
			__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = ELEMENT(T,b,i,localIndexY+j,pitch2);
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
			}
			ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch3) = sum;
		}
	}
}

// Умножение двух матриц с использованием global памяти
template<class T> __global__ void __global__matrix_mul__constant__none__(T *c, int height, int width_height, int width, size_t pitch) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			T sum = 0;
			for (int k = 0; k < width_height ; k++ ) {
				sum += ELEMENT(T,__c__,i,k,width_height*sizeof(T)) * ELEMENT(T,__c__,k,j+IDX(height,0,width_height),width*sizeof(T));
			}
			ELEMENT(T,c,i,j,pitch) = sum;
		}
	}
}

// Умножение двух матриц с использованием shared памяти
// Using shared memory
// • Similar to local memory:
// – __shared__ float current_row[];
// • Only declare one variable as shared!
// – Multiple declarations of __shared__
// variables will occupy same memory space!
// – __shared__ float a[];
// – __shared__ float b[];
// – b[0] = 0.5f;
// – now a[0] == 0.5f also!
template<class T> __global__ void __global__matrix_mul__constant__shared__(T *c, int height,int width_height, int width, size_t pitch) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	
	__shared__ T __s__[SHARED_SIZE];
	
	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
			__s__[IDX(i,j,width_height)] = ELEMENT(T,__c__,sharedIndexX+i,j,width_height*sizeof(T));
		}
	}

	for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = ELEMENT(T,__c__,i,sharedIndexY+j+IDX(height,0,width_height),width*sizeof(T));
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
			}
			ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch) = sum;
		}
	}
}

// Умножение двух матриц с использованием local памяти
// Local memory
// • Also read/write per-thread
// • Can’t read other threads’ local memory
// – Declare a variable in local memory using
// the __local__ keyword
// – __local__ float results[32];
// • Can index (this is where local arrays go)
// • Much slower than register memory!
// – Don’t use local arrays if you don’t have to
template<class T> __global__ void __global__matrix_mul__constant__local__(T *c, int height,int width_height, int width, size_t pitch) {
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x);
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y);
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x)-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/blockDim.x/gridDim.x);
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y)-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/blockDim.y/gridDim.y);
	
	T __l__[LOCAL_SIZE];
	
	for (int i = 0; i < localCountX; i ++) {
		for (int j = 0; j < width_height; j++) {
			__l__[IDX(i,j,width_height)] = ELEMENT(T,__c__,localIndexX+i,j, width_height*sizeof(T));
		}
	}

	for (int i = 0; i < width_height; i ++) {
		for (int j = 0; j < localCountY; j++) {
			__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = ELEMENT(T,__c__,i,localIndexY+j+IDX(height,0,width_height), width*sizeof(T));
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
			}
			ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch) = sum;
		}
	}
}

// Умножение двух матриц с использованием texture памяти
template<class T> __global__ void __global__matrix_mul__texture__none__(T *c, int height, int width_height, int width, size_t pitch) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			T sum = 0;
			for (int k = 0; k < width_height ; k++ ) {
				uint4 buffer_a = tex1Dfetch(tex_a,IDX(i,k,width_height));
				uint4 buffer_b = tex1Dfetch(tex_b,IDX(k,j,width));
				sum += *(T*)&buffer_a + *(T*)&buffer_b;
			}
			ELEMENT(T,c,i,j,pitch) = sum;
		}
	}
}

// Умножение двух матриц с использованием shared памяти
// Using shared memory
// • Similar to local memory:
// – __shared__ float current_row[];
// • Only declare one variable as shared!
// – Multiple declarations of __shared__
// variables will occupy same memory space!
// – __shared__ float a[];
// – __shared__ float b[];
// – b[0] = 0.5f;
// – now a[0] == 0.5f also!
template<class T> __global__ void __global__matrix_mul__texture__shared__(T *c, int height,int width_height, int width, size_t pitch) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	
	__shared__ T __s__[SHARED_SIZE];
	
	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
			uint4 buffer_a = tex1Dfetch(tex_a,IDX(sharedIndexX+i,j,width_height));
			__s__[IDX(i,j,width_height)] = *(T*)&buffer_a;
		}
	}

	for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			uint4 buffer_b = tex1Dfetch(tex_b,IDX(i,sharedIndexY+j,width));
			__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = *(T*)&buffer_b;
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
			}
			ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch) = sum;
		}
	}
}

// Умножение двух матриц с использованием local памяти
// Local memory
// • Also read/write per-thread
// • Can’t read other threads’ local memory
// – Declare a variable in local memory using
// the __local__ keyword
// – __local__ float results[32];
// • Can index (this is where local arrays go)
// • Much slower than register memory!
// – Don’t use local arrays if you don’t have to
template<class T> __global__ void __global__matrix_mul__texture__local__(T *c, int height,int width_height, int width, size_t pitch) {
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x);
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y);
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/blockDim.x/gridDim.x)-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/blockDim.x/gridDim.x);
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/blockDim.y/gridDim.y)-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/blockDim.y/gridDim.y);
	
	T __l__[LOCAL_SIZE];
	
	for (int i = 0; i < localCountX; i ++) {
		for (int j = 0; j < width_height; j++) {
			uint4 buffer_a = tex1Dfetch(tex_a,IDX(localIndexX+i,j,width_height));
			__l__[IDX(i,j,width_height)] = *(T*)&buffer_a;
		}
	}

	for (int i = 0; i < width_height; i ++) {
		for (int j = 0; j < localCountY; j++) {
			uint4 buffer_b = tex1Dfetch(tex_b,IDX(i,localIndexY+j,width));
			__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = *(T*)&buffer_b;
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			T sum = 0;
			for(int k=0; k<width_height; k++) {
				sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
			}
			ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch) = sum;
		}
	}
}

template<class T> __host__ void __host__matrix_mul(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, MATRIX<T> *c, MEMORY src, MEMORY dest, MEMORY cache) {
	T * d_a;
	T * d_b;
	T * d_c;
	size_t pitch1;
	size_t pitch2;
	size_t pitch3;
	hipError_t err;

	int height = a->height;
	int width_height = a->width;
	int width =  b->width;
	c->height = height;
	c->width = width;

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipHostMalloc((void**)&cpu_b, (size_t) b->width * b->height * sizeof(uint4));
		err = hipMalloc((void**)&gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipMalloc((void**)&gpu_b, (size_t) b->width * b->height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				uint4 buffer;
				*(T*)&buffer = a->values[IDX(i,j,width)];
				cpu_a[IDX(i,j,width)] = buffer;
			}
		}
		for(int i=0; i<b->height; i++) {
			for(int j=0; j<b->width; j++) {
				uint4 buffer;
				*(T*)&buffer = b->values[IDX(i,j,width)];
				cpu_b[IDX(i,j,width)] = buffer;
			}
		}
		//  настройка параемтров текстуры  texture
		tex_a.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_a.addressMode[1] = hipAddressModeWrap;
		tex_a.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_a.normalized     = false;                // не использовать нормализованную адресацию
		tex_b.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_b.addressMode[1] = hipAddressModeWrap;
		tex_b.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_b.normalized     = false;                // не использовать нормализованную адресацию
		err = hipBindTexture(0, tex_a, gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipBindTexture(0, tex_b, gpu_b, (size_t) b->width * b->height * sizeof(uint4));
		err = hipMemcpy((void*)gpu_a, (void*)cpu_a, (size_t) a->width * a->height * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipMemcpy((void*)gpu_b, (void*)cpu_b, (size_t) b->width * b->height * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipHostFree((void*)cpu_a);
		err = hipHostFree((void*)cpu_b);
		break;
	case GLOBAL:
		err = hipMallocPitch((void**)&d_a, &pitch1, (size_t) a->width * sizeof(T), (size_t) a->height);
		err = hipMallocPitch((void**)&d_b, &pitch2, (size_t) b->width * sizeof(T), (size_t) b->height);
		err = hipMemcpy2D((void*)d_a, pitch1, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		err = hipMemcpy2D((void*)d_b, pitch2, (void*)b->values, (size_t) b->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		break;
	case CONSTANT:
		err = hipMemcpyToSymbol(HIP_SYMBOL(__c__), (void*)a->values, (size_t) a->height * a->width * sizeof(T), (size_t) 0, hipMemcpyHostToDevice);
		err = hipMemcpyToSymbol(HIP_SYMBOL(__c__), (void*)b->values, (size_t) b->height * b->width * sizeof(T), (size_t) height*width_height*sizeof(T), hipMemcpyHostToDevice);
		break;
	default:
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMallocPitch((void**)&d_c, &pitch3, (size_t) c->width * sizeof(T), (size_t) c->height);
		break;
	default:
		break;
	}

	if (src == GLOBAL && dest == GLOBAL && cache == NONE) {
		__global__matrix_mul__global__none__<T><<<blocks,threads>>>(d_a, d_b, d_c, height, width_height, width, pitch1, pitch2, pitch3);
	}
	else if (src == GLOBAL && dest == GLOBAL && cache == LOCAL) {
		__global__matrix_mul__global__local__<T><<<blocks,threads>>>(d_a, d_b, d_c, height, width_height, width, pitch1, pitch2, pitch3);
	}
	else if (src == GLOBAL && dest == GLOBAL && cache == SHARED) {
		__global__matrix_mul__global__shared__<T><<<blocks,threads>>>(d_a, d_b, d_c, height, width_height, width, pitch1, pitch2, pitch3);
	}
	else if (src == CONSTANT && dest == GLOBAL && cache == NONE) {
		__global__matrix_mul__constant__none__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}
	else if (src == CONSTANT && dest == GLOBAL && cache == LOCAL) {
		__global__matrix_mul__constant__local__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}
	else if (src == CONSTANT && dest == GLOBAL && cache == SHARED) {
		__global__matrix_mul__constant__shared__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}
	else if (src == TEXTURE && dest == GLOBAL && cache == NONE) {
		__global__matrix_mul__texture__none__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}
	else if (src == TEXTURE && dest == GLOBAL && cache == LOCAL) {
		__global__matrix_mul__texture__local__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}
	else if (src == TEXTURE && dest == GLOBAL && cache == SHARED) {
		__global__matrix_mul__texture__shared__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3);
	}

	switch(src) {
	case TEXTURE:
		err = hipUnbindTexture(tex_a);
		err = hipUnbindTexture(tex_b);
		err = hipFree(gpu_a);
		err = hipFree(gpu_b);
		break;
	case CONSTANT:
		break;
	default:
		err = hipFree(d_a);
		err = hipFree(d_b);
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMemcpy2D((void*)c->values, (size_t) c->width * sizeof(T), (void*)d_c, pitch3, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
		err = hipFree(d_c);
		break;
	default:
		break;
	}
	
	err = err;
}


