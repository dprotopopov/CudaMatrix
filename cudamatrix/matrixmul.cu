#include "hip/hip_runtime.h"
﻿#include "matrix.h"
#include "globals.h"

// Умножение двух матриц с использованием global памяти
template<class T> __global__ void __global__matrix_mul__global__(T *a, T *b, T *c, int height, int width_height, int width, size_t pitch1, size_t pitch2, size_t pitch3, MEMORY cache) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));

	T __l__[LOCAL_SIZE];
	__shared__ T __s__[SHARED_SIZE];

	switch(cache){
	case NONE:
		for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
			for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
				T sum = 0;
				for (int k = 0; k < width_height ; k++ ) {
					sum += ELEMENT(T,a,i,k,pitch1) * ELEMENT(T,b,k,j,pitch2);
				}
				ELEMENT(T,c,i,j,pitch3) = sum;
			}
		}
		break;
	case SHARED:
		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
				__s__[IDX(i,j,width_height)] = ELEMENT(T,a,sharedIndexX+i,j,pitch1);
			}
		}

		for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
				__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = ELEMENT(T,b,i,sharedIndexY+j,pitch2);
			}
		}

		__syncthreads();

		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
				}
				ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch3) = sum;
			}
		}
		break;
	case LOCAL:
		for (int i = 0; i < localCountX; i ++) {
			for (int j = 0; j < width_height; j++) {
				__l__[IDX(i,j,width_height)] = ELEMENT(T,a,localIndexX+i,j,pitch1);
			}
		}

		for (int i = 0; i < width_height; i ++) {
			for (int j = 0; j < localCountY; j++) {
				__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = ELEMENT(T,b,i,localIndexY+j,pitch2);
			}
		}

		for (int i = 0; i < localCountX; i++ ) {
			for (int j = 0; j < localCountY; j++ ) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
				}
				ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch3) = sum;
			}
		}
		break;
	}
}


// Умножение двух матриц с использованием constant памяти
template<class T> __global__ void __global__matrix_mul__constant__(T *c, int height, int width_height, int width, size_t pitch, MEMORY cache) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));

	T __l__[LOCAL_SIZE];
	__shared__ T __s__[SHARED_SIZE];

	switch(cache){
	case NONE:
		for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
			for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
				T sum = 0;
				for (int k = 0; k < width_height ; k++ ) {
					buffer_a.i = __c__[IDX(i,k,width_height)];
					buffer_b.i = __c__[IDX(k,j,width)+IDX(height,0,width_height)];
					sum +=  buffer_a.t * buffer_b.t;
				}
				ELEMENT(T,c,i,j,pitch) = sum;
			}
		}
		break;
	case SHARED:
		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
				buffer_a.i = __c__[IDX(sharedIndexX+i,j,width_height)];
				__s__[IDX(i,j,width_height)] = buffer_a.t;
			}
		}

		for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
				buffer_b.i = __c__[IDX(i,sharedIndexY+j,width)+IDX(height,0,width_height)];
				__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = buffer_b.t;
			}
		}

		__syncthreads();

		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
				}
				ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch) = sum;
			}
		}
		break;
	case LOCAL:
		for (int i = 0; i < localCountX; i ++) {
			for (int j = 0; j < width_height; j++) {
				buffer_a.i = __c__[IDX(localIndexX+i,j,width_height)];
				__l__[IDX(i,j,width_height)] = buffer_a.t;
			}
		}

		for (int i = 0; i < width_height; i ++) {
			for (int j = 0; j < localCountY; j++) {
				buffer_b.i = __c__[IDX(i,localIndexY+j,width)+IDX(height,0,width_height)];
				__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = buffer_b.t;
			}
		}

		for (int i = 0; i < localCountX; i++ ) {
			for (int j = 0; j < localCountY; j++ ) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
				}
				ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch) = sum;
			}
		}
		break;
	}
}


// Умножение двух матриц с использованием texture памяти
template<class T> __global__ void __global__matrix_mul__texture__(T *c, int height, int width_height, int width, size_t pitch, MEMORY cache) {
	texturestorage<T> buffer_a;
	texturestorage<T> buffer_b;
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));

	T __l__[LOCAL_SIZE];
	__shared__ T __s__[SHARED_SIZE];

	switch(cache){
	case NONE:
		for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
			for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
				T sum = 0;
				for (int k = 0; k < width_height ; k++ ) {
					buffer_a.i = tex1Dfetch(tex_a,IDX(i,k,width_height));
					buffer_b.i = tex1Dfetch(tex_b,IDX(k,j,width));
					sum += buffer_a.t * buffer_b.t;
				}
				ELEMENT(T,c,i,j,pitch) = sum;
			}
		}
		break;
	case SHARED:
		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < width_height; j+= blockDim.y) {
				buffer_a.i = tex1Dfetch(tex_a,IDX(sharedIndexX+i,j,width_height));
				__s__[IDX(i,j,width_height)] = buffer_a.t;
			}
		}

		for (int i = threadIdx.x; i < width_height; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
				buffer_b.i = tex1Dfetch(tex_b,IDX(i,sharedIndexY+j,width));
				__s__[SHARED_SIZE - IDX(i,j,sharedCountY) - 1] = buffer_b.t;
			}
		}

		__syncthreads();

		for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
			for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __s__[IDX(i,k,width_height)] * __s__[SHARED_SIZE - IDX(k,j,sharedCountY) - 1];
				}
				ELEMENT(T,c,sharedIndexX+i,sharedIndexY+j,pitch) = sum;
			}
		}
		break;
	case LOCAL:
		for (int i = 0; i < localCountX; i ++) {
			for (int j = 0; j < width_height; j++) {
				buffer_a.i = tex1Dfetch(tex_a,IDX(localIndexX+i,j,width_height));
				__l__[IDX(i,j,width_height)] = buffer_a.t;
			}
		}

		for (int i = 0; i < width_height; i ++) {
			for (int j = 0; j < localCountY; j++) {
				buffer_b.i = tex1Dfetch(tex_b,IDX(i,localIndexY+j,width));
				__l__[LOCAL_SIZE - IDX(i,j,localCountY) - 1] = buffer_b.t;
			}
		}

		for (int i = 0; i < localCountX; i++ ) {
			for (int j = 0; j < localCountY; j++ ) {
				T sum = 0;
				for(int k=0; k<width_height; k++) {
					sum += __l__[IDX(i,k,width_height)] * __l__[LOCAL_SIZE - IDX(k,j,localCountY) - 1];
				}
				ELEMENT(T,c,localIndexX+i,localIndexY+j,pitch) = sum;
			}
		}
		break;
	}
}

template<class T> __host__ void __cdecl __host__matrix_mul(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, MATRIX<T> *c, MEMORY src, MEMORY dest, MEMORY cache) {
	T * h_a;
	T * h_b;
	T * d_a;
	T * d_b;
	T * d_c;
	size_t pitch1;
	size_t pitch2;
	size_t pitch3;
	hipError_t err;
	storage<T> buffer;
	texturestorage<T> texturebuffer;

	int height = a->height;
	int width_height = a->width;
	int width =  b->width;
	c->height = height;
	c->width = width;

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_a, (size_t) a->width * a->height * sizeof(TEXTURESTORAGETYPE));
		err = hipHostMalloc((void**)&cpu_b, (size_t) b->width * b->height * sizeof(TEXTURESTORAGETYPE));
		err = hipMalloc((void**)&gpu_a, (size_t) a->width * a->height * sizeof(TEXTURESTORAGETYPE));
		err = hipMalloc((void**)&gpu_b, (size_t) b->width * b->height * sizeof(TEXTURESTORAGETYPE));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				texturebuffer.t = a->values[IDX(i,j,width_height)];
				cpu_a[IDX(i,j,width_height)] = texturebuffer.i;
			}
		}
		for(int i=0; i<b->height; i++) {
			for(int j=0; j<b->width; j++) {
				texturebuffer.t = b->values[IDX(i,j,width)];
				cpu_b[IDX(i,j,width)] = texturebuffer.i;
			}
		}
		//  настройка параемтров текстуры  texture
		tex_a.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_a.addressMode[1] = hipAddressModeWrap;
		tex_a.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_a.normalized     = false;                // не использовать нормализованную адресацию
		tex_b.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_b.addressMode[1] = hipAddressModeWrap;
		tex_b.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_b.normalized     = false;                // не использовать нормализованную адресацию
		err = hipMemcpy((void*)gpu_a, (void*)cpu_a, (size_t) a->width * a->height * sizeof(TEXTURESTORAGETYPE), hipMemcpyHostToDevice);
		err = hipMemcpy((void*)gpu_b, (void*)cpu_b, (size_t) b->width * b->height * sizeof(TEXTURESTORAGETYPE), hipMemcpyHostToDevice);
		err = hipBindTexture(0, tex_a, gpu_a, (size_t) a->width * a->height * sizeof(TEXTURESTORAGETYPE));
		err = hipBindTexture(0, tex_b, gpu_b, (size_t) b->width * b->height * sizeof(TEXTURESTORAGETYPE));
		err = hipHostFree((void*)cpu_a);
		err = hipHostFree((void*)cpu_b);
		break;
	case GLOBAL:
		err = hipMallocPitch((void**)&d_a, &pitch1, (size_t) a->width * sizeof(T), (size_t) a->height);
		err = hipMallocPitch((void**)&d_b, &pitch2, (size_t) b->width * sizeof(T), (size_t) b->height);
		err = hipMemcpy2D((void*)d_a, pitch1, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width_height * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		err = hipMemcpy2D((void*)d_b, pitch2, (void*)b->values, (size_t) b->width * sizeof(T), (size_t) width * sizeof(T), (size_t) width_height, hipMemcpyHostToDevice);
		break;
	case CONSTANT:
		err = hipHostMalloc((void**)&h_a, (size_t) a->width * a->height * sizeof(STORAGETYPE));
		err = hipHostMalloc((void**)&h_b, (size_t) b->width * b->height * sizeof(STORAGETYPE));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width_height)];
				h_a[IDX(i,j,width_height)] = buffer.i;
			}
		}
		for(int i=0; i<b->height; i++) {
			for(int j=0; j<b->width; j++) {
				buffer.t = b->values[IDX(i,j,width)];
				h_b[IDX(i,j,width)] = buffer.i;
			}
		}
		err = hipGetSymbolAddress((void **)&d_a, __c__);
		d_b=&d_a[height*width_height];
		err = hipMemcpy(d_a, h_a, (size_t) height * width_height * sizeof(STORAGETYPE), hipMemcpyHostToDevice);		
		err = hipMemcpy(d_b, h_b, (size_t) width_height * width * sizeof(STORAGETYPE), hipMemcpyHostToDevice);		
		err = hipHostFree((void*)h_a);
		err = hipHostFree((void*)h_b);
		break;
	default:
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMallocPitch((void**)&d_c, &pitch3, (size_t) c->width * sizeof(T), (size_t) c->height);
		break;
	default:
		break;
	}

	if (src == GLOBAL && dest == GLOBAL) {
		__global__matrix_mul__global__<T><<<blocks,threads>>>(d_a, d_b, d_c, height, width_height, width, pitch1, pitch2, pitch3, cache);
	}
	else if (src == CONSTANT && dest == GLOBAL) {
		__global__matrix_mul__constant__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3, cache);
	}
	else if (src == TEXTURE && dest == GLOBAL) {
		__global__matrix_mul__texture__<T><<<blocks,threads>>>(d_c, height, width_height, width, pitch3, cache);
	}

	switch(src) {
	case TEXTURE:
		err = hipUnbindTexture(tex_a);
		err = hipUnbindTexture(tex_b);
		err = hipFree(gpu_a);
		err = hipFree(gpu_b);
		break;
	case CONSTANT:
		break;
	default:
		err = hipFree(d_a);
		err = hipFree(d_b);
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMemcpy2D((void*)c->values, (size_t) c->width * sizeof(T), (void*)d_c, pitch3, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
		err = hipFree(d_c);
		break;
	default:
		break;
	}
	
	err = err;
}

template __host__ void __cdecl __host__matrix_mul<DATATYPE>(dim3 blocks, dim3 threads, MATRIX<DATATYPE> *a, MATRIX<DATATYPE> *b, MATRIX<DATATYPE> *c, MEMORY src, MEMORY dest, MEMORY cache);

