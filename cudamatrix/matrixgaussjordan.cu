#include "hip/hip_runtime.h"
﻿#include "matrix.h"
#include "globals.h"

template<class T> __global__ void __global__matrix_gaussjordanstep__global__none__(T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	T d = ELEMENT(T,prev,row,col,pitch);
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			if (i!=row && j!=col) {
				ELEMENT(T,next,i,j,pitch) = ELEMENT(T,prev,i,j,pitch) - ELEMENT(T,prev,i,col,pitch)*ELEMENT(T,prev,row,j,pitch)/ d;
			}
			else if (i==row && j!=col) {
				ELEMENT(T,next,i,j,pitch) = ELEMENT(T,prev,i,j,pitch) / d;
			}
			else if (i!=row && j==col) {
				ELEMENT(T,next,i,j,pitch) = (T)0;
			}
			else {
				ELEMENT(T,next,i,j,pitch) = (T)1;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep2__global__none__(T *buffer, T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	T d = buffer[row];
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			if (i!=row) {
				ELEMENT(T,next,i,j,pitch) = ELEMENT(T,prev,i,j,pitch) - buffer[i]*ELEMENT(T,prev,row,j,pitch)/ d;
			}
			else {
				ELEMENT(T,next,i,j,pitch) = ELEMENT(T,prev,i,j,pitch) / d;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep__global__shared__(T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);

	T d = ELEMENT(T,prev,row,col,pitch);

	__shared__ T __s__[SHARED_SIZE];

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < 1; j+= blockDim.y) {
			__s__[i] = ELEMENT(T,prev,sharedIndexX+i,col,pitch);
		}
	}
	for (int i = threadIdx.x; i < 1; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			__s__[SHARED_SIZE - j - 1] = ELEMENT(T,prev,row,sharedIndexY+j,pitch);
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
			if (sharedIndexX+i!=row && sharedIndexY+j!=col) {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = ELEMENT(T,prev,sharedIndexX+i,sharedIndexY+j,pitch) - __s__[i]*__s__[SHARED_SIZE - j - 1]/ d;
			}
			else if (sharedIndexX+i==row && sharedIndexY+j!=col) {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = __s__[SHARED_SIZE - j - 1] /d;
			}
			else if (sharedIndexX+i!=row && sharedIndexY+j==col) {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = (T)0;
			}
			else {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = (T)1;
			}
		}
	}
}
template<class T> __global__ void __global__matrix_gaussjordanstep2__global__shared__(T *buffer, T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);

	T d = buffer[row];

	__shared__ T __s__[SHARED_SIZE];

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < 1; j+= blockDim.y) {
			__s__[i] = buffer[sharedIndexX+i];
		}
	}
	for (int i = threadIdx.x; i < 1; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			__s__[SHARED_SIZE - j - 1] = ELEMENT(T,prev,row,sharedIndexY+j,pitch);
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
			if (sharedIndexX+i!=row) {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = ELEMENT(T,prev,sharedIndexX+i,sharedIndexY+j,pitch) - __s__[i]*__s__[SHARED_SIZE - j - 1]/ d;
			}
			else {
				ELEMENT(T,next,sharedIndexX+i,sharedIndexY+j,pitch) = __s__[SHARED_SIZE - j - 1] / d;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep__global__local__(T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y));

	T d = ELEMENT(T,prev,row,col,pitch);

	T __l__[LOCAL_SIZE];

	for (int i = 0; i < localCountX; i++) {
		for (int j = 0; j < 1; j++) {
			__l__[i] = ELEMENT(T,prev,localIndexX+i,col,pitch);
		}
	}
	for (int i = 0; i < 1; i++) {
		for (int j = 0; j < localCountY; j++) {
			__l__[LOCAL_SIZE - j - 1] = ELEMENT(T,prev,row,localIndexY+j,pitch);
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			if (localIndexX+i!=row && localIndexY+j!=col) {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = ELEMENT(T,prev,localIndexX+i,localIndexY+j,pitch) - __l__[i]*__l__[LOCAL_SIZE - j - 1]/ d;
			}
			else if (localIndexX+i==row && localIndexY+j!=col) {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = __l__[LOCAL_SIZE - j - 1] / d;
			}
			else if (localIndexX+i!=row && localIndexY+j==col) {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = (T)0;
			}
			else {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = (T)1;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep2__global__local__(T *buffer, T *prev, T *next, int row, int col, int height, int width, size_t pitch) {
	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y));

	T d = buffer[row];

	T __l__[LOCAL_SIZE];

	for (int i = 0; i < localCountX; i++) {
		for (int j = 0; j < 1; j++) {
			__l__[i] = buffer[localIndexX+i];
		}
	}
	for (int i = 0; i < 1; i++) {
		for (int j = 0; j < localCountY; j++) {
			__l__[LOCAL_SIZE - j - 1] = ELEMENT(T,prev,row,localIndexY+j,pitch);
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			if (localIndexX+i!=row) {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = ELEMENT(T,prev,localIndexX+i,localIndexY+j,pitch) - __l__[i]*__l__[LOCAL_SIZE - j - 1]/ d;
			}
			else {
				ELEMENT(T,next,localIndexX+i,localIndexY+j,pitch) = __l__[LOCAL_SIZE - j - 1] / d;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep__texture__none__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;
	buffer_d.i = tex1Dfetch(tex_w,IDX(row,col,width));
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			if (i!=row && j!=col) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(i,j,width));
				buffer_b.i = tex1Dfetch(tex_w,IDX(i,col,width));
				buffer_c.i = tex1Dfetch(tex_w,IDX(row,j,width));
				buffer_e.t =  buffer_a.t - buffer_b.t*buffer_c.t/ buffer_d.t;
				next[IDX(i,j,width)] = buffer_e.i;
			}
			else if (i==row && j!=col) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(i,j,width));
				buffer_e.t = buffer_a.t / buffer_d.t;
				next[IDX(i,j,width)] = buffer_e.i;
			}
			else if (i!=row && j==col) {
				buffer_e.t = (T)0;
				next[IDX(i,j,width)] = buffer_e.i;
			}
			else {
				buffer_e.t = (T)1;
				next[IDX(i,j,width)] = buffer_e.i;
			}
		}
	}
}
template<class T> __global__ void __global__matrix_gaussjordanstep2__texture__none__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;
	buffer_d.i = tex1Dfetch(tex_buffer,row);
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			if (i!=row) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(i,j,width));
				buffer_b.i = tex1Dfetch(tex_buffer,i);
				buffer_c.i = tex1Dfetch(tex_w,IDX(row,j,width));
				buffer_e.t =  buffer_a.t - buffer_b.t*buffer_c.t/ buffer_d.t;
				next[IDX(i,j,width)] = buffer_e.i;
			}
			else {
				buffer_a.i = tex1Dfetch(tex_w,IDX(i,j,width));
				buffer_e.t = buffer_a.t / buffer_d.t;
				next[IDX(i,j,width)] = buffer_e.i;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep__texture__shared__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;

	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);

	buffer_d.i = tex1Dfetch(tex_w,IDX(row,col,width));

	__shared__ T __s__[SHARED_SIZE];

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < 1; j+= blockDim.y) {
			buffer_b.i = tex1Dfetch(tex_w,IDX(sharedIndexX+i,col,width));
			__s__[i] = buffer_b.t;
		}
	}
	for (int i = threadIdx.x; i < 1; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			buffer_c.i = tex1Dfetch(tex_w,IDX(row,sharedIndexY+j,width));
			__s__[SHARED_SIZE - j - 1] = buffer_c.t;
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
			if (sharedIndexX+i!=row && sharedIndexY+j!=col) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(sharedIndexX+i,sharedIndexY+j,width));
				buffer_e.t = buffer_a.t - __s__[i]*__s__[SHARED_SIZE - j - 1]/ buffer_d.t;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
			else if (sharedIndexX+i==row && sharedIndexY+j!=col) {
				buffer_e.t = __s__[SHARED_SIZE - j - 1] / buffer_d.t;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
			else if (sharedIndexX+i!=row && sharedIndexY+j==col) {
				buffer_e.t = (T)0;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
			else {
				buffer_e.t = (T)1;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep2__texture__shared__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;

	int sharedIndexX = (int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedIndexY = (int)((width*(blockIdx.y+0))/gridDim.y);
	int sharedCountX = (int)((height*(blockIdx.x+1))/gridDim.x)-(int)((height*(blockIdx.x+0))/gridDim.x);
	int sharedCountY = (int)((width*(blockIdx.y+1))/gridDim.y)-(int)((width*(blockIdx.y+0))/gridDim.y);

	buffer_d.i = tex1Dfetch(tex_buffer,row);

	__shared__ T __s__[SHARED_SIZE];

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < 1; j+= blockDim.y) {
			buffer_b.i = tex1Dfetch(tex_buffer,sharedIndexX+i);
			__s__[i] = buffer_b.t;
		}
	}
	for (int i = threadIdx.x; i < 1; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j+= blockDim.y) {
			buffer_c.i = tex1Dfetch(tex_w,IDX(row,sharedIndexY+j,width));
			__s__[SHARED_SIZE - j - 1] = buffer_c.t;
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < sharedCountX; i += blockDim.x) {
		for (int j = threadIdx.y; j < sharedCountY; j += blockDim.y) {
			if (sharedIndexX+i!=row) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(sharedIndexX+i,sharedIndexY+j,width));
				buffer_e.t = buffer_a.t - __s__[i]*__s__[SHARED_SIZE - j - 1]/ buffer_d.t;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
			else {
				buffer_e.t = __s__[SHARED_SIZE - j - 1] / buffer_d.t;
				next[IDX(sharedIndexX+i,sharedIndexY+j,width)] = buffer_e.i;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep__texture__local__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;

	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y));

	buffer_d.i = tex1Dfetch(tex_w,IDX(row,col,width));

	T __l__[LOCAL_SIZE];

	for (int i = 0; i < localCountX; i++) {
		for (int j = 0; j < 1; j++) {
			buffer_b.i = tex1Dfetch(tex_w,IDX(localIndexX+i,col,width));
			__l__[i] = buffer_b.t;
		}
	}
	for (int i = 0; i < 1; i++) {
		for (int j = 0; j < localCountY; j++) {
			buffer_c.i = tex1Dfetch(tex_w,IDX(row,localIndexY+j,width));
			__l__[LOCAL_SIZE - j - 1] = buffer_c.t;
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			if (localIndexX+i!=row && localIndexY+j!=col) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(localIndexX+i,localIndexY+j,width));
				buffer_e.t = buffer_a.t - __l__[i]*__l__[LOCAL_SIZE - j - 1]/ buffer_d.t;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
			else if (localIndexX+i==row && localIndexY+j!=col) {
				buffer_e.t = __l__[LOCAL_SIZE - j - 1] / buffer_d.t;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
			else if (localIndexX+i!=row && localIndexY+j==col) {
				buffer_e.t = (T)0;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
			else {
				buffer_e.t = (T)1;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_gaussjordanstep2__texture__local__(uint4 *next, int row, int col, int height, int width) {
	storage<T> buffer_a;
	storage<T> buffer_b;
	storage<T> buffer_c;
	storage<T> buffer_d;
	storage<T> buffer_e;

	int localIndexX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x));
	int localIndexY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y));
	int localCountX = (int)((height*(blockDim.x*blockIdx.x+threadIdx.x+0))/(blockDim.x*gridDim.x))-(int)((height*(blockDim.x*blockIdx.x+threadIdx.x+1))/(blockDim.x*gridDim.x));
	int localCountY = (int)((width*(blockDim.y*blockIdx.y+threadIdx.y+0))/(blockDim.y*gridDim.y))-(int)((width*(blockDim.y*blockIdx.y+threadIdx.y+1))/(blockDim.y*gridDim.y));

	buffer_d.i = tex1Dfetch(tex_buffer,row);

	T __l__[LOCAL_SIZE];

	for (int i = 0; i < localCountX; i++) {
		for (int j = 0; j < 1; j++) {
			buffer_b.i = tex1Dfetch(tex_buffer,localIndexX+i);
			__l__[i] = buffer_b.t;
		}
	}
	for (int i = 0; i < 1; i++) {
		for (int j = 0; j < localCountY; j++) {
			buffer_c.i = tex1Dfetch(tex_w,IDX(row,localIndexY+j,width));
			__l__[LOCAL_SIZE - j - 1] = buffer_c.t;
		}
	}

	for (int i = 0; i < localCountX; i++ ) {
		for (int j = 0; j < localCountY; j++ ) {
			if (localIndexX+i!=row) {
				buffer_a.i = tex1Dfetch(tex_w,IDX(localIndexX+i,localIndexY+j,width));
				buffer_e.t = buffer_a.t - __l__[i]*__l__[LOCAL_SIZE - j - 1]/ buffer_d.t;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
			else {
				buffer_e.t = __l__[LOCAL_SIZE - j - 1] / buffer_d.t;
				next[IDX(localIndexX+i,localIndexY+j,width)] = buffer_e.i;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_find_first_notzero__global__(T *a, int *b, int height, int width, T tolerance, size_t pitch) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		b[i] = -1;
		for(int index = 0;index<width;index++) {
			T x = ELEMENT(T,a,i,index,pitch);
			if (abs(x)>-tolerance) {
				b[i] = index;
				break;
			}
		}
	}
}

template<class T> __global__ void __global__matrix_find_first_notzero__texture__(int *b, int height, int width, T tolerance) {
	storage<T> buffer;
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		b[i] = -1;
		for(int index = 0;index<width;index++) {
			buffer.i = tex1Dfetch(tex_w,IDX(i,index,width));
			if (abs(buffer.t)>-tolerance) {
				b[i] = index;
				break;
			}
		}
	}
}

template<class T> __host__ void __cdecl __host__matrix_gaussjordanstep(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, int row, int col, MEMORY src, MEMORY dest, MEMORY cache) {
	T *d_w[2];
	size_t pitch;
	hipError_t err;
	storage<T> buffer;

	int height = a->height;
	int width =  a->width;
	b->height = height;
	b->width = width;

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_wv, (size_t) width * height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width)];
				cpu_wv[IDX(i,j,width)] = buffer.i;
			}
		}
		for(int i=0; i<2; i++) {
			err = hipMalloc((void**)&gpu_w[i], (size_t) width * height * sizeof(uint4));
		}
		//  настройка параемтров текстуры  texture
		tex_w.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_w.addressMode[1] = hipAddressModeWrap;
		tex_w.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_w.normalized     = false;                // не использовать нормализованную адресацию
		err = hipMemcpy((void*)gpu_w[0], (void*)cpu_wv, (size_t) width * height * sizeof(uint4), hipMemcpyHostToDevice);
		break;
	default:
		for(int i=0; i<2; i++) {
			err = hipMallocPitch((void**)&d_w[i], &pitch, (size_t) width * sizeof(T), (size_t) height);
		}
		err = hipMemcpy2D((void*)d_w[0], pitch, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		break;
	}

	if(src == GLOBAL && cache == NONE) {
		__global__matrix_gaussjordanstep__global__none__<T><<<blocks,threads>>>(d_w[0],d_w[1], row, col, height, width, pitch);
	}
	else if(src == GLOBAL && cache == SHARED) {
		__global__matrix_gaussjordanstep__global__shared__<T><<<blocks,threads>>>(d_w[0],d_w[1], row, col, height, width, pitch);
	}
	else if(src == GLOBAL && cache == LOCAL) {
		__global__matrix_gaussjordanstep__global__local__<T><<<blocks,threads>>>(d_w[0],d_w[1], row, col, height, width, pitch);
	}
	else if(src == TEXTURE && cache == NONE) {
		err = hipBindTexture(0, tex_w, gpu_w[0], (size_t) width * height * sizeof(uint4));
		__global__matrix_gaussjordanstep__texture__none__<T><<<blocks,threads>>>(gpu_w[1], row, col, height, width);
		err = hipUnbindTexture(tex_w);
	}
	else if(src == TEXTURE && cache == SHARED) {
		err = hipBindTexture(0, tex_w, gpu_w[0], (size_t) width * height * sizeof(uint4));
		__global__matrix_gaussjordanstep__texture__shared__<T><<<blocks,threads>>>(gpu_w[1], row, col, height, width);
		err = hipUnbindTexture(tex_w);
	}
	else if(src == TEXTURE && cache == LOCAL) {
		err = hipBindTexture(0, tex_w, gpu_w[0], (size_t) width * height * sizeof(uint4));
		__global__matrix_gaussjordanstep__texture__local__<T><<<blocks,threads>>>(gpu_w[1], row, col, height, width);
		err = hipUnbindTexture(tex_w);
	}

	switch(src) {
	case TEXTURE:
		err = hipMemcpy((void*)cpu_wv, (void*)gpu_w[1], (size_t) width * height * sizeof(uint4), hipMemcpyDeviceToHost);
		for(int i=0; i<height; i++) {
			for(int j=0; j<width; j++) {
				b->values[IDX(i,j,width)] = *(T*)&cpu_wv[IDX(i,j,width)];
			}
		}
		for(int i=0; i<2; i++) {
			err = hipFree(gpu_w[i]);
		}
		err = hipHostFree((void*)cpu_wv);
		break;
	default:
		err = hipMemcpy2D((void*)b->values, (size_t) b->width * sizeof(T), (void*)d_w[1], pitch, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
		for(int i=0; i<2; i++) {
			err = hipFree(d_w[i]);
		}
		break;
	}

	err = err;
}

template<class T> __host__ void __cdecl __host__matrix_gaussjordan(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, T tolerance, MEMORY src, MEMORY dest, MEMORY cache) {
	T *d_w[2];
	int *h_index;
	int *d_index;
	size_t pitch;
	hipError_t err;
	storage<T> buffer;

	int height = a->height;
	int width =  a->width;
	b->height = height;
	b->width = width;

	err = hipHostMalloc((void**)&h_index, height*sizeof(int));
	err = hipMalloc((void**)&d_index, height*sizeof(int));

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_wv, (size_t) width * height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width)];
				cpu_wv[IDX(i,j,width)] = buffer.i;
			}
		}
		for(int i=0; i<2; i++) {
			err = hipMalloc((void**)&gpu_w[i], (size_t) width * height * sizeof(uint4));
		}
		//  настройка параемтров текстуры  texture
		tex_w.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_w.addressMode[1] = hipAddressModeWrap;
		tex_w.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_w.normalized     = false;                // не использовать нормализованную адресацию
		err = hipMemcpy((void*)gpu_w[0], (void*)cpu_wv, (size_t) width * height * sizeof(uint4), hipMemcpyHostToDevice);
		break;
	default:
		for(int i=0; i<2; i++) {
			err = hipMallocPitch((void**)&d_w[i], &pitch, (size_t) width * sizeof(T), (size_t) height);
		}
		err = hipMemcpy2D((void*)d_w[0], pitch, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		break;
	}
	
	for(int k=0;;k++) {
		if(src == GLOBAL) {
			__global__matrix_find_first_notzero__global__<T><<<1,height>>>(d_w[k&1],d_index,height,width,tolerance,pitch);
		}
		else if(src == TEXTURE) {
			err = hipBindTexture(0, tex_w, gpu_w[k&1], (size_t) width * height * sizeof(uint4));
			__global__matrix_find_first_notzero__texture__<T><<<1,height>>>(d_index,height,width,tolerance);
		}
		hipMemcpy((void*)h_index,(void*)d_index,(size_t) height*sizeof(int),hipMemcpyDeviceToHost);
		while(h_index[k]<0 && k<min(height,width)) k++;
		if(k == min(height,width)) {
			switch(src) {
			case TEXTURE:
				err = hipUnbindTexture(tex_w);
				err = hipMemcpy((void*)cpu_wv, (void*)gpu_w[k&1], (size_t) width * height * sizeof(uint4), hipMemcpyDeviceToHost);
				for(int i=0; i<height; i++) {
					for(int j=0; j<width; j++) {
						b->values[IDX(i,j,width)] = *(T*)&cpu_wv[IDX(i,j,width)];
					}
				}
				break;
			case GLOBAL:
			default:
				err = hipMemcpy2D((void*)b->values, (size_t) b->width * sizeof(T), (void*)d_w[k&1], pitch, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
				break;
			}
			break;
		}
		if(src == GLOBAL && cache == NONE) {
			__global__matrix_gaussjordanstep__global__none__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], height, width, pitch);
		}
		else if(src == GLOBAL && cache == SHARED) {
			__global__matrix_gaussjordanstep__global__shared__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], height, width, pitch);
		}
		else if(src == GLOBAL && cache == LOCAL) {
			__global__matrix_gaussjordanstep__global__local__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], height, width, pitch);
		}
		else if(src == TEXTURE && cache == NONE) {
			__global__matrix_gaussjordanstep__texture__none__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], height, width);
			err = hipUnbindTexture(tex_w);
		}
		else if(src == TEXTURE && cache == SHARED) {
			__global__matrix_gaussjordanstep__texture__shared__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], height, width);
			err = hipUnbindTexture(tex_w);
		}
		else if(src == TEXTURE && cache == LOCAL) {
			__global__matrix_gaussjordanstep__texture__local__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], height, width);
			err = hipUnbindTexture(tex_w);
		}
	}

	switch(src) {
	case TEXTURE:
		for(int i=0; i<2; i++) {
			err = hipFree(gpu_w[i]);
		}
		err = hipHostFree((void*)cpu_wv);
		break;
	default:
		for(int i=0; i<2; i++) {
			err = hipFree(d_w[i]);
		}
		break;
	}

	err = hipFree(d_index);
	err = hipHostFree((void*)h_index);

	err = err;
}

// Вычисление обратной матрицы
template<class T> __host__ void __cdecl __host__matrix_inv(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, T tolerance,  MEMORY src, MEMORY dest, MEMORY cache) {
	T *d_w[2];
	T *d_v[2];
	int *h_index;
	int *d_index;
	size_t pitch;
	T * d_buffer;
	hipError_t err;
	storage<T> buffer;

	int height = a->height;
	int width =  a->width;
	b->height = height;
	b->width = width;
	
	assert(a->height == a->width);

	if (height==1 && width==1) {
		b->values[0] = (T)1/a->values[0];
		return;
	}

	err = hipHostMalloc((void**)&h_index, (size_t) width * sizeof(int));
	err = hipMalloc((void**)&d_index, (size_t) width * sizeof(int));

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_wv, (size_t) width * width * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width)];
				cpu_wv[IDX(i,j,width)] = buffer.i;
			}
		}
		for(int i=0; i<2; i++) {
			err = hipMalloc((void**)&gpu_w[i], (size_t) width * width * sizeof(uint4));
			err = hipMalloc((void**)&gpu_v[i], (size_t) width * width * sizeof(uint4));
		}
		//  настройка параемтров текстуры  texture
		tex_w.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_w.addressMode[1] = hipAddressModeWrap;
		tex_w.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_w.normalized     = false;                // не использовать нормализованную адресацию
		err = hipMemcpy((void*)gpu_w[0], (void*)cpu_wv, (size_t) width * width * sizeof(uint4), hipMemcpyHostToDevice);
		//  настройка параемтров текстуры  texture
		tex_v.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_v.addressMode[1] = hipAddressModeWrap;
		tex_v.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_v.normalized     = false;                // не использовать нормализованную адресацию
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = (i==j)?(T)1:(T)0;
				cpu_wv[IDX(i,j,width)] = buffer.i;
			}
		}
		err = hipMemcpy((void*)gpu_v[0], (void*)cpu_wv, (size_t) width * width * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipMalloc((void**)&gpu_buffer, (size_t) width * sizeof(uint4));
		//  настройка параемтров текстуры  texture
		tex_buffer.addressMode[0] = hipAddressModeWrap;  // режим Wrap
		tex_buffer.addressMode[1] = hipAddressModeWrap;
		tex_buffer.filterMode     = hipFilterModePoint;  // ближайшее значение
		tex_buffer.normalized     = false;                // не использовать нормализованную адресацию
		err = hipBindTexture(0, tex_buffer, gpu_buffer, (size_t) width * width * sizeof(uint4));
		break;
	default:
		for(int i=0; i<2; i++) {
			err = hipMallocPitch((void**)&d_w[i], &pitch, (size_t) width * sizeof(T), (size_t) width);
			err = hipMallocPitch((void**)&d_v[i], &pitch, (size_t) width * sizeof(T), (size_t) width);
		}
		err = hipMemcpy2D((void*)d_w[0], pitch, (void*)a->values, (size_t) width * sizeof(T), (size_t) width * sizeof(T), width, hipMemcpyHostToDevice);
		for(int i=0; i<height; i++) {
			for(int j=0; j<width; j++) {
				b->values[IDX(i,j,width)] = (i==j)?(T)1:(T)0;
			}
		}
		err = hipMemcpy2D((void*)d_v[0], pitch, (void*)b->values, (size_t) width * sizeof(T), (size_t) width * sizeof(T), width, hipMemcpyHostToDevice);
		err = hipMalloc((void**)&d_buffer, (size_t) width * sizeof(T));
		break;
	}

	for(int k=0;;k++) {
		if(src == GLOBAL) {
			__global__matrix_find_first_notzero__global__<T><<<1,width>>>(d_w[k&1],d_index,width,width,tolerance,pitch);
		}
		else if(src == TEXTURE) {
			err = hipBindTexture(0, tex_w, gpu_w[k&1], (size_t) width * width * sizeof(uint4));
			__global__matrix_find_first_notzero__texture__<T><<<1,width>>>(d_index,width,width,tolerance);
		}
		err = hipMemcpy((void*)h_index,(void*)d_index,(size_t) width * sizeof(int),hipMemcpyDeviceToHost);
		while(h_index[k]<0 && k<min(height,width)) k++;
		if(k == min(height,width)) {
			switch(src) {
			case GLOBAL:
				err = hipMemcpy2D((void*)b->values,width*sizeof(T),(void*)d_v[k&1],pitch,width*sizeof(T),width,hipMemcpyDeviceToHost);
				break;
			case TEXTURE:
				err = hipUnbindTexture(tex_w);
				err = hipMemcpy((void*)cpu_wv,(void*)gpu_v[k&1],width*width*sizeof(uint4),hipMemcpyDeviceToHost);
				for(int i=0; i<height; i++) {
					for(int j=0; j<width; j++) {
						buffer.i=cpu_wv[IDX(i,j,width)];
						b->values[IDX(i,j,width)] = buffer.t;
					}
				}
				break;
			}
			break;
		}
		if(src == GLOBAL && cache == NONE) {
			err = hipMemcpy2D((void*)d_buffer, 1*sizeof(T), (void*)&ELEMENT(T,d_w[k&1],0,h_index[k],pitch), pitch, (size_t) 1 * sizeof(T), width, hipMemcpyDeviceToDevice);
			__global__matrix_gaussjordanstep__global__none__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], width, width, pitch);
			__global__matrix_gaussjordanstep2__global__none__<T><<<blocks,threads>>>(d_buffer,d_v[k&1],d_v[1-(k&1)], k, h_index[k], width, width, pitch);
		}
		else if(src == GLOBAL && cache == SHARED) {
			err = hipMemcpy2D((void*)d_buffer, 1*sizeof(T), (void*)&ELEMENT(T,d_w[k&1],0,h_index[k],pitch), pitch, (size_t) 1 * sizeof(T), width, hipMemcpyDeviceToDevice);
			__global__matrix_gaussjordanstep__global__shared__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], width, width, pitch);
			__global__matrix_gaussjordanstep2__global__shared__<T><<<blocks,threads>>>(d_buffer,d_v[k&1],d_v[1-(k&1)], k, h_index[k], width, width, pitch);
		}
		else if(src == GLOBAL && cache == LOCAL) {
			err = hipMemcpy2D((void*)d_buffer, 1*sizeof(T), (void*)&ELEMENT(T,d_w[k&1],0,h_index[k],pitch), pitch, (size_t) 1 * sizeof(T), width, hipMemcpyDeviceToDevice);
			__global__matrix_gaussjordanstep__global__local__<T><<<blocks,threads>>>(d_w[k&1],d_w[1-(k&1)], k, h_index[k], width, width, pitch);
			__global__matrix_gaussjordanstep2__global__local__<T><<<blocks,threads>>>(d_buffer,d_v[k&1],d_v[1-(k&1)], k, h_index[k], width, width, pitch);
		}
		else if(src == TEXTURE && cache == NONE) {
			err = hipMemcpy2D((void*)gpu_buffer, 1*sizeof(uint4), (void*)&gpu_w[k&1][IDX(0,h_index[k],width)], width*sizeof(uint4), (size_t) 1 * sizeof(uint4), width, hipMemcpyDeviceToDevice);
			err = hipBindTexture(0, tex_v, gpu_v[k&1], (size_t) width * width * sizeof(uint4));
			__global__matrix_gaussjordanstep__texture__none__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], width, width);
			__global__matrix_gaussjordanstep2__texture__none__<T><<<blocks,threads>>>(gpu_v[1-(k&1)], k, h_index[k], width, width);
			err = hipUnbindTexture(tex_w);
			err = hipUnbindTexture(tex_v);
		}
		else if(src == TEXTURE && cache == SHARED) {
			err = hipMemcpy2D((void*)gpu_buffer, 1*sizeof(uint4), (void*)&gpu_w[k&1][IDX(0,h_index[k],width)], width*sizeof(uint4), (size_t) 1 * sizeof(uint4), width, hipMemcpyDeviceToDevice);
			err = hipBindTexture(0, tex_v, gpu_v[k&1], (size_t) width * width * sizeof(uint4));
			__global__matrix_gaussjordanstep__texture__shared__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], width, width);
			__global__matrix_gaussjordanstep2__texture__shared__<T><<<blocks,threads>>>(gpu_v[1-(k&1)], k, h_index[k], width, width);
			err = hipUnbindTexture(tex_w);
			err = hipUnbindTexture(tex_v);
		}
		else if(src == TEXTURE && cache == LOCAL) {
			err = hipMemcpy2D((void*)gpu_buffer, 1*sizeof(uint4), (void*)&gpu_w[k&1][IDX(0,h_index[k],width)], width*sizeof(uint4), (size_t) 1 * sizeof(uint4), width, hipMemcpyDeviceToDevice);
			err = hipBindTexture(0, tex_v, gpu_v[k&1], (size_t) width * width * sizeof(uint4));
			__global__matrix_gaussjordanstep__texture__local__<T><<<blocks,threads>>>(gpu_w[1-(k&1)], k, h_index[k], width, width);
			__global__matrix_gaussjordanstep2__texture__local__<T><<<blocks,threads>>>(gpu_v[1-(k&1)], k, h_index[k], width, width);
			err = hipUnbindTexture(tex_w);
			err = hipUnbindTexture(tex_v);
		}
	}
	
	switch(src) {
	case TEXTURE:
		for(int i=0; i<2; i++) {
			err = hipFree(gpu_w[i]);
			err = hipFree(gpu_v[i]);
		}
		err = hipFree(cpu_wv);
		err = hipFree(gpu_buffer);
		break;
	default:
		for(int i=0; i<2; i++) {
			err = hipFree(d_w[i]);
			err = hipFree(d_v[i]);
		}
		err = hipFree(d_buffer);
		break;
	}

	err = hipFree(d_index);
	err = hipHostFree(h_index);

	err = err;
}

template __host__ void __cdecl __host__matrix_gaussjordanstep<double>(dim3 blocks, dim3 threads, MATRIX<double> *a, MATRIX<double> *b, int row, int col, MEMORY src, MEMORY dest, MEMORY cache);
template __host__ void __cdecl __host__matrix_gaussjordan<double>(dim3 blocks, dim3 threads, MATRIX<double> *a, MATRIX<double> *b, double tolerance, MEMORY src, MEMORY dest, MEMORY cache);
template __host__ void __cdecl __host__matrix_inv<double>(dim3 blocks, dim3 threads, MATRIX<double> *a, MATRIX<double> *b, double tolerance, MEMORY src, MEMORY dest, MEMORY cache);
