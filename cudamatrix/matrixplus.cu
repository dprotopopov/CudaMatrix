#include "hip/hip_runtime.h"
#include "matrix.h"
#include "globals.h"

// ������� ���� ������� � �������������� global ������
template<class T> __global__ void __global__matrix_plus__global__(T *a, T *b, int height, int width, size_t pitch1, size_t pitch2) {
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			ELEMENT(T,b,i,j,pitch2) = ELEMENT(T,a,i,j,pitch1);
		}
	}
}

// ������� ���� ������� � �������������� constant ������
template<class T> __global__ void __global__matrix_plus__constant__(T *b, int height, int width, size_t pitch) {
	storage<T> buffer_a;
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			buffer_a.i = __c__[IDX(i,j,width)];
			ELEMENT(T,b,i,j,pitch) = buffer_a.t;
		}
	}
}

// ������� ���� ������� � �������������� texture ������
template<class T> __global__ void __global__matrix_plus__texture__(T *b, int height, int width, size_t pitch) {
	storage<T> buffer_a;
	for (int i = blockDim.x*blockIdx.x + threadIdx.x; i < height; i += blockDim.x*gridDim.x) {
		for (int j = blockDim.y*blockIdx.y + threadIdx.y; j < width; j += blockDim.y*gridDim.y) {
			buffer_a.i = tex1Dfetch(tex_a,IDX(i,j,width));
			ELEMENT(T,b,i,j,pitch) = buffer_a.t;
		}
	}
}

template<class T> __host__ void __cdecl __host__matrix_plus(dim3 blocks, dim3 threads, MATRIX<T> *a, MATRIX<T> *b, MEMORY src, MEMORY dest, MEMORY cache) {

	T * d_a;
	T * d_b;
	size_t pitch1;
	size_t pitch2;
	hipError_t err;
	storage<T> buffer;

	int height = a->height;
	int width =  a->width;
	b->height = height;
	b->width = width;

	switch(src) {
	case TEXTURE:
		err = hipHostMalloc((void**)&cpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipMalloc((void**)&gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width)];
				cpu_a[IDX(i,j,width)] = buffer.i;
			}
		}
		//  ��������� ���������� ��������  texture
		tex_a.addressMode[0] = hipAddressModeWrap;  // ����� Wrap
		tex_a.addressMode[1] = hipAddressModeWrap;
		tex_a.filterMode     = hipFilterModePoint;  // ��������� ��������
		tex_a.normalized     = false;                // �� ������������ ��������������� ���������
		err = hipMemcpy((void*)gpu_a, (void*)cpu_a, (size_t) a->width * a->height * sizeof(uint4), hipMemcpyHostToDevice);
		err = hipBindTexture(0, tex_a, gpu_a, (size_t) a->width * a->height * sizeof(uint4));
		err = hipHostFree((void*)cpu_a);
		break;
	case GLOBAL:
		err = hipMallocPitch((void**)&d_a, &pitch1, (size_t) a->width * sizeof(T), (size_t) a->height);
		err = hipMemcpy2D((void*)d_a, pitch1, (void*)a->values, (size_t) a->width * sizeof(T), (size_t) width * sizeof(T), (size_t) height, hipMemcpyHostToDevice);
		break;
	case CONSTANT:
		err = hipHostMalloc((void**)&cpu_a, (size_t) a->width * a->height * sizeof(uint4));
		for(int i=0; i<a->height; i++) {
			for(int j=0; j<a->width; j++) {
				buffer.t = a->values[IDX(i,j,width)];
				cpu_a[IDX(i,j,width)] = buffer.i;
			}
		}
		err = hipGetSymbolAddress((void **)&gpu_a, __c__);
		err = hipMemcpy(gpu_a, cpu_a, (size_t) height * width * sizeof(uint4), hipMemcpyHostToDevice);		
		err = hipHostFree((void*)cpu_a);
		break;
	default:
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMallocPitch((void**)&d_b, &pitch2, (size_t) b->width * sizeof(T), (size_t) b->height);
		break;
	default:
		break;
	}

	if (src == GLOBAL && dest == GLOBAL) {
		__global__matrix_plus__global__<T><<<blocks,threads>>>(d_a, d_b, height, width, pitch1, pitch2);
	}
	else if (src == CONSTANT && dest == GLOBAL) {
		__global__matrix_plus__constant__<T><<<blocks,threads>>>(d_b, height, width, pitch2);
	}
	else if (src == TEXTURE && dest == GLOBAL) {
		__global__matrix_plus__texture__<T><<<blocks,threads>>>(d_b, height, width, pitch2);
	}

	switch(src) {
	case TEXTURE:
		err = hipUnbindTexture(tex_a);
		err = hipFree(gpu_a);
		break;
	case CONSTANT:
		break;
	default:
		hipFree(d_a);
		break;
	}
	switch(dest) {
	case GLOBAL:
		err = hipMemcpy2D((void*)b->values, (size_t) b->width * sizeof(T), (void*)d_b, pitch2, (size_t) width * sizeof(T), (size_t) height, hipMemcpyDeviceToHost);
		err = hipFree(d_b);
		break;
	default:
		break;
	}
	
	err = err;
}

template __host__ void __cdecl __host__matrix_plus<double>(dim3 blocks, dim3 threads, MATRIX<double> *a, MATRIX<double> *b, MEMORY src, MEMORY dest, MEMORY cache);
