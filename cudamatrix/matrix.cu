#include "hip/hip_runtime.h"
﻿#define __NO_EXTERN__
#include "matrix.h"

// Constant memory (64KB max) can only be hard-coded before compilation.
__constant__ char __cdecl __c__[CONST_SIZE];

texture<uint4, hipTextureType1D, hipReadModeElementType> __cdecl tex_a;
texture<uint4, hipTextureType1D, hipReadModeElementType> __cdecl tex_b;
texture<uint4, hipTextureType1D, hipReadModeElementType> __cdecl tex_w;
texture<uint4, hipTextureType1D, hipReadModeElementType> __cdecl tex_v;
texture<uint4, hipTextureType1D, hipReadModeElementType> __cdecl tex_buffer;

uint4 * __cdecl gpu_a;
uint4 * __cdecl gpu_b;
uint4 * __cdecl cpu_a;
uint4 * __cdecl cpu_b;
uint4 * __cdecl gpu_w[2];
uint4 * __cdecl gpu_v[2];
uint4 * __cdecl cpu_wv;
uint4 * __cdecl gpu_buffer;

char * __cdecl string_stack[1024];
int __cdecl string_stack_size = 0;

double __cdecl double_stack[1024];
int __cdecl double_stack_size = 0;

